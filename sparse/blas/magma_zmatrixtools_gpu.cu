#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"

#define PRECISION_z

#define SWAP(a, b)  { tmp = a; a = b; b = tmp; }



__global__ void 
magma_zvalinit_kernel(  
    const magma_int_t num_el, 
    magmaDoubleComplex_ptr dval) 
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    if (k < num_el) {
        dval[k] = zero;
    }
}


/**
    Purpose
    -------
    
    Initializes a device array with zero. 

    Arguments
    ---------

    @param[in]
    num_el      magma_int_t
                size of array

    @param[in,out]
    dval        magmaDoubleComplex_ptr
                array to initialize
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.
                
    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zvalinit_gpu(
    magma_int_t num_el,
    magmaDoubleComplex_ptr dval,
    magma_queue_t queue)
{
    int blocksize1 = 128;
    int blocksize2 = 1;

    int dimgrid1 = magma_ceildiv(num_el, blocksize1);
    int dimgrid2 = 1;
    int dimgrid3 = 1;
    dim3 grid(dimgrid1, dimgrid2, dimgrid3);
    dim3 block(blocksize1, blocksize2, 1);
    magma_zvalinit_kernel<<< grid, block, 0, queue->hip_stream() >>>
        (num_el, dval);

    return MAGMA_SUCCESS;
}




__global__ void 
magma_zindexinit_kernel(  
    const magma_int_t num_el, 
    magmaIndex_ptr dind) 
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_el) {
        dind[k] = 0;
    }
}


/**
    Purpose
    -------
    
    Initializes a device array with zero. 

    Arguments
    ---------

    @param[in]
    num_el      magma_int_t
                size of array

    @param[in,out]
    dind        magmaIndex_ptr
                array to initialize
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.
                
    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zindexinit_gpu(
    magma_int_t num_el,
    magmaIndex_ptr dind,
    magma_queue_t queue)
{
    int blocksize1 = 128;
    int blocksize2 = 1;

    int dimgrid1 = magma_ceildiv(num_el, blocksize1);
    int dimgrid2 = 1;
    int dimgrid3 = 1;
    dim3 grid(dimgrid1, dimgrid2, dimgrid3);
    dim3 block(blocksize1, blocksize2, 1);
    magma_zindexinit_kernel<<< grid, block, 0, queue->hip_stream() >>>
        (num_el, dind);

    return MAGMA_SUCCESS;
}


/***************************************************************************//**
    Purpose
    -------
    Generates a matrix $U = A \cup B$. If both matrices have a nonzero value
    in the same location, the value of A is used.
    
    This is the GPU version of the operation.

    Arguments
    ---------

    @param[in]
    A           magma_z_matrix
                Input matrix 1.

    @param[in]
    B           magma_z_matrix
                Input matrix 2.

    @param[out]
    U           magma_z_matrix*
                $U = A \cup B$. If both matrices have a nonzero value
                in the same location, the value of A is used.

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zaux
*******************************************************************************/

extern "C" magma_int_t
magma_zcsr_sort_gpu(
    magma_z_matrix *A,
    magma_queue_t queue)
{   
    magma_int_t info = 0;
    hipsparseHandle_t handle=NULL;
    hipsparseMatDescr_t descrA=NULL;
    
    magmaDoubleComplex_ptr tmp=NULL, csrVal_sorted=NULL;
    char* pBuffer;
    int *P;
    size_t pBufferSizeInBytes;
    
    CHECK_CUSPARSE( hipsparseCreate( &handle ));
    CHECK_CUSPARSE( hipsparseSetStream( handle, queue->hip_stream() ));
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &descrA ));
    CHECK_CUSPARSE( hipsparseSetMatType( descrA, 
        HIPSPARSE_MATRIX_TYPE_GENERAL ));
    CHECK_CUSPARSE( hipsparseSetMatDiagType( descrA, 
        HIPSPARSE_DIAG_TYPE_NON_UNIT ));
    CHECK_CUSPARSE( hipsparseSetMatIndexBase( descrA, 
        HIPSPARSE_INDEX_BASE_ZERO ));
    
    CHECK(magma_zmalloc(&csrVal_sorted, A->nnz));
   
    // step 1: allocate buffer
    hipsparseXcsrsort_bufferSizeExt(handle, A->num_rows, A->num_cols, 
        A->nnz, A->drow, A->dcol, &pBufferSizeInBytes);
    hipMalloc( &pBuffer, sizeof(char)* pBufferSizeInBytes);
    
    // step 2: setup permutation vector P to identity
    hipMalloc( (void**)&P, sizeof(int)*A->nnz);
    hipsparseCreateIdentityPermutation(handle, A->nnz, P);
    
    // step 3: sort CSR format
    hipsparseXcsrsort(handle, A->num_rows, A->num_cols, A->nnz, 
        descrA, A->drow, A->dcol, P, pBuffer);
    
    // step 4: gather sorted csrVal
#if CUDA_VERSION >= 12000
    hipsparseSpVecDescr_t vec_permutation;
    hipsparseDnVecDescr_t vec_values;
    CHECK_CUSPARSE( hipsparseCreateSpVec(&vec_permutation, A->nnz, A->nnz,
                                        P, csrVal_sorted,
                                        HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vec_values, A->nnz, A->dval, HIP_C_64F) );
    CHECK_CUSPARSE( hipsparseGather(handle, vec_values, vec_permutation) );
    
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpVec(vec_permutation) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vec_values) );
#else
    hipsparseZgthr(handle, A->nnz, (hipDoubleComplex*)A->dval, (hipDoubleComplex*)csrVal_sorted, P, 
        HIPSPARSE_INDEX_BASE_ZERO);
#endif

    SWAP(A->dval, csrVal_sorted);
    
cleanup:
    hipsparseDestroyMatDescr( descrA );
    hipsparseDestroy( handle );
    magma_free(csrVal_sorted);

    return info;
}
